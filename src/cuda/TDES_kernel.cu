#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <string>
#include <sstream>
#include <cstdlib>

__global__
void encode(char* in, char* out,unsigned int size)
{
    uint index = blockIdx.x*blockDim.x + threadIdx.x;
	if(index>=size)
	{
		return;
	}
	
	// Parse hex block into 64-bit
	uint64_t block = *((uint64_t*)(in+16*index));//std::stoull(message.substr(16 * i, 16), 0, 16);
	// Encode with k1, decode with k2, encode with k3
	uint64_t blockPass1 = this->processBlock(block, 0, false);
	uint64_t blockPass2 = this->processBlock(blockPass1, 1, true);
	uint64_t blockPass3 = this->processBlock(blockPass2, 2, false);

	// Return as hex string
	std::stringstream hexString;
	hexString << std::hex << std::setfill('0') << std::setw(16) << blockPass3;
	hipMemcpy(out+32*i,hexString.str().z_str(),32,hipMemcpyDeviceToDevice);
}

__global__
void decode(char* in, char* out,unsigned int size)
{
	uint index = blockIdx.x*blockDim.x + threadIdx.x;
	if(index>=size)
	{
		return;
	}
	
	// Parse hex block into 64-bit
	uint64_t block = *((uint64_t*)(in+16*index));//std::stoull(message.substr(16 * i, 16), 0, 16);
	// Decode with k3, encode with k2, decode with k1
	uint64_t blockPass1 = this->processBlock(block, 2, true);
	uint64_t blockPass2 = this->processBlock(blockPass1, 1, false);
	uint64_t blockPass3 = this->processBlock(blockPass2, 0, true);

	// Return as hex string
	std::stringstream hexString;
	hexString << std::hex << std::setfill('0') << std::setw(16) << blockPass3;
	hipMemcpy(out+32*i,hexString.str().z_str(),32,hipMemcpyDeviceToDevice);
}